
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel() {
  printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
  printf("Hello from the CPU\n");

  hello_kernel<<<1, 10>>>();

  hipDeviceReset();

  return 0;
}
